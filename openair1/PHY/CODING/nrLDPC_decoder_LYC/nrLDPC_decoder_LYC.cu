#include "hip/hip_runtime.h"
 /*! \file PHY/CODING/nrLDPC_decoder_LYC/nrLDPC_decoder_LYC.cu
 * \brief LDPC cuda support BG1 all length
 * \author NCTU OpinConnect Terng-Yin Hsu,WEI-YING,LIN
 * \email tyhsu@cs.nctu.edu.tw
 * \date 13-05-2020
 * \version 
 * \note
 * \warning
 */
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "PHY/CODING/nrLDPC_decoder/nrLDPC_types.h"
#include "PHY/CODING/nrLDPC_decoder/nrLDPCdecoder_defs.h"
#include "assertions.h"
#include "bgs/BG1_I0"
#include "bgs/BG1_I1"
#include "bgs/BG1_I2"
#include "bgs/BG1_I3"
#include "bgs/BG1_I4"
#include "bgs/BG1_I5"
#include "bgs/BG1_I6"
#include "bgs/BG1_I7"
#include "bgs/BG2_I0"
#include "bgs/BG2_I1"
#include "bgs/BG2_I2"
#include "bgs/BG2_I3"
#include "bgs/BG2_I4"
#include "bgs/BG2_I5"
#include "bgs/BG2_I6"
#include "bgs/BG2_I7"

#define MAX_ITERATION 2
#define MC	1

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}

typedef struct{
  char x;
  char y;
  short value;
} h_element;
#include "bgs/BG1_compact_in_C.h"

__device__ char dev_const_llr[68*384];
__device__ char dev_dt [46*68*384];
__device__ char dev_llr[68*384];
__device__ unsigned char dev_tmp[68*384];

h_element h_compact1 [46*23] = {};
h_element h_compact2 [68*30] = {};

__device__  h_element dev_h_compact1[46*23];  // used in kernel 1
__device__  h_element dev_h_compact2[68*30];  // used in kernel 2

// __device__ __constant__ h_element dev_h_compact1[46*23];  // used in kernel 1
// __device__ __constant__ h_element dev_h_compact2[68*30];  // used in kernel 2

// row and col element count
__device__ __constant__ char h_ele_row_bg1_count[46] = {
	19, 19, 19, 19, 3,  8,  9,  7, 10,  9,
	7,  8,  7,  6,  7,  7,  6,  6,  6,  6,  
	6,  6,  5,  5,  6,  5,  5,  4,  5,  5,  
	5,  5,  5,  5,  5,  5,  5,  4,  5,  5,  
	4,  5,  4,  5,  5,  4};
__device__ __constant__ char h_ele_col_bg1_count[68] = {
	30, 28,  7, 11,  9,  4,  8, 12,  8,  7, 
	12, 10, 12, 11, 10,  7, 10, 10, 13,  7,  
	8,  11, 12,  5,  6,  6,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1};
__device__ __constant__ char h_ele_row_bg2_count[42] = { 
	8, 10,  8, 10,  4,  6,  6,  6,  4,  5, 
	5,  5,  4,  5,  5,  4,  5,  5,  4,  4,
	4,  4,  3,  4,  4,  3,  5,  3,  4,  3,
	5,  3,  4,  4,  4,  4,  4,  3,  4,  4, 
	4,  4};
__device__ __constant__ char h_ele_col_bg2_count[52] = {
	22, 23, 10,  5,  5, 14,  7, 13,  6,  8, 
	9,  16,  9, 12,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1, 
	1,   1};


__global__ void warmup()
{
	// warm up gpu for time measurement
}

extern "C"
void warmup_for_GPU(){
	
	warmup<<<20,1024 >>>();

}

extern "C"
void set_compact_BG(int Zc,short BG){
	
	int row,col;
	if(BG == 1){
		row = 46;
		col = 68;
	}
	else{
		row = 42;
		col = 52;
	}
	int compact_row = 30; 
	int compact_col = 19;
	if(BG==2){compact_row = 10, compact_col = 23;}
	int memorySize_h_compact1 = row * compact_col * sizeof(h_element);
	int memorySize_h_compact2 = compact_row * col * sizeof(h_element);
	int lift_index = 0;
	short lift_set[][9] = {
		{2,4,8,16,32,64,128,256},
		{3,6,12,24,48,96,192,384},
		{5,10,20,40,80,160,320},
		{7,14,28,56,112,224},
		{9,18,36,72,144,288},
		{11,22,44,88,176,352},
		{13,26,52,104,208},
		{15,30,60,120,240},
		{0}
	};
	
	for(int i = 0; lift_set[i][0] != 0; i++){
		for(int j = 0; lift_set[i][j] != 0; j++){
			if(Zc == lift_set[i][j]){
				lift_index = i;
				break;
			}
		}
	}
	printf("\nZc = %d BG = %d\n",Zc,BG);
	switch(lift_index){
			case 0:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I0, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I0, memorySize_h_compact2) );
				break;
			case 1:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I1, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I1, memorySize_h_compact2) );
				break;
			case 2:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I2, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I2, memorySize_h_compact2) );
				break;
			case 3:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I3, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I3, memorySize_h_compact2) );
				break;
			case 4:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I4, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I4, memorySize_h_compact2) );
				break;
			case 5:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I5, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I5, memorySize_h_compact2) );
				break;
			case 6:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I6, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I6, memorySize_h_compact2) );
				break;
			case 7:
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact1), host_h_compact1_I7, memorySize_h_compact1) );
				cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_h_compact2), host_h_compact2_I7, memorySize_h_compact2) );
				break;
		}
	
	// return 0;
}


// Kernel 1
__global__ void ldpc_cnp_kernel_1st_iter(/*char * dev_llr,*/ int BG, int row, int col, int Zc)
{
//	if(blockIdx.x == 0 && threadIdx.x == 1) printf("cnp %d\n", threadIdx.x);
	int iMCW = blockIdx.y;		// codeword id
	int iBlkRow = blockIdx.x;	// block row in h_base
	int iBlkCol;				// block col in h_base
	int iSubRow = threadIdx.x;	// row index in sub_block of h_base
	int iCol;					// overall col index in h_base
	int offsetR;
	int shift_t;

//	For 2-min algorithm.
	int Q_sign = 0;
	int sq;
	int Q, Q_abs;
	int R_temp;

	int sign = 1;
	int rmin1 = INT32_MAX;
	int rmin2 = INT32_MAX;
	char idx_min = 0;

	h_element h_element_t;
	int s = (BG==1)? h_ele_row_bg1_count[iBlkRow]:h_ele_row_bg2_count[iBlkRow];
	offsetR = (iMCW * row*col*Zc) + iBlkRow * Zc + iSubRow;	// row*col*Zc = size of dev_dt
//	if(blockIdx.x == 0 && threadIdx.x == 1) printf("s: %d, offset %d\n", s, offsetR);
//	The 1st recursion
	for(int i = 0; i < s; i++) // loop through all the ZxZ sub-blocks in a row
	{
		h_element_t = dev_h_compact1[i*row+iBlkRow];	// compact_col == row

		iBlkCol = h_element_t.y;
		shift_t = h_element_t.value;

		shift_t = (iSubRow + shift_t) % Zc;
		iCol = (iMCW * col*Zc) + iBlkCol * Zc + shift_t;	// col*Zc = size of llr
		Q = dev_llr[iCol];
		Q_abs = (Q>0)? Q : -Q;
		sq = Q < 0;
//		if(blockIdx.x == 0 && threadIdx.x == 1) printf("i %d, icol %d, Q: %d\n", i, iCol, Q);
		// quick version
		sign = sign * (1 - sq * 2);
		Q_sign |= sq << i;

		if (Q_abs < rmin1){
			rmin2 = rmin1;
			rmin1 = Q_abs;
			idx_min = i;
		} else if (Q_abs < rmin2){
			rmin2 = Q_abs;
		}
	}

//	if(blockIdx.x == 0 && threadIdx.x == 1)printf("min1 %d, min2 %d, min1_idx %d\n", rmin1, rmin2, idx_min);

//	The 2nd recursion
	for(int i = 0; i < s; i++){
		// v0: Best performance so far. 0.75f is the value of alpha.
		sq = 1 - 2 * ((Q_sign >> i) & 0x01);
		R_temp = 0.75f * sign * sq * (i != idx_min ? rmin1 : rmin2);
		// write results to global memory
		h_element_t = dev_h_compact1[i*row+iBlkRow];
		int addr_temp = offsetR + h_element_t.y * row * Zc;
		dev_dt[addr_temp] = R_temp;
//		if(blockIdx.x == 0 && threadIdx.x == 1)printf("R_temp %d, temp_addr %d\n", R_temp, addr_temp);
	}	
}

// Kernel_1
__global__ void ldpc_cnp_kernel(/*char * dev_llr, char * dev_dt,*/ int BG, int row, int col, int Zc)
{
//	if(blockIdx.x == 0 && threadIdx.x == 1) printf("cnp\n");
	int iMCW = blockIdx.y;
	int iBlkRow = blockIdx.x;	// block row in h_base
	int iBlkCol; 				// block col in h_base
	int iSubRow = threadIdx.x;	// row index in sub_block of h_base
	int iCol; 					// overall col index in h_base
	int offsetR;
	int shift_t;

//	For 2-min algorithm.
	int Q_sign = 0;
	int sq;
	int Q, Q_abs;
	int R_temp;

	int sign = 1;
	int rmin1 = INT32_MAX;
	int rmin2 = INT32_MAX;
	char idx_min = 0;

	h_element h_element_t;
	int s = (BG==1)? h_ele_row_bg1_count[iBlkRow]: h_ele_row_bg2_count[iBlkRow];
	offsetR = (iMCW *row*col*Zc) + iBlkRow * Zc + iSubRow;	// row * col * Zc = size of dev_dt
//	if(blockIdx.x == 0 && threadIdx.x == 1) printf("s: %d, offset %d\n", s, offsetR);
//	The 1st recursion
	for(int i = 0; i < s; i++) // loop through all the ZxZ sub-blocks in a row
	{
		h_element_t = dev_h_compact1[i*row+iBlkRow];

		iBlkCol = h_element_t.y;
		shift_t = h_element_t.value;
		shift_t = (iSubRow + shift_t) % Zc;
		iCol = iBlkCol * Zc + shift_t;
		
		R_temp = dev_dt[offsetR + iBlkCol * row * Zc];

		Q = dev_llr[iMCW * (col*Zc) + iCol] - R_temp;
		Q_abs = (Q>0)? Q : -Q;
//		if(blockIdx.x == 0 && threadIdx.x == 1) printf("i %d, icol %d, Q: %d\n", i, iCol, Q);
		sq = Q < 0;
		sign = sign * (1 - sq * 2);
		Q_sign |= sq << i;

		if (Q_abs < rmin1){
			rmin2 = rmin1;
			rmin1 = Q_abs;
			idx_min = i;
		} else if (Q_abs < rmin2){
			rmin2 = Q_abs;
		}
		
	}

//	if(blockIdx.x == 0 && threadIdx.x == 1)printf("min1 %d, min2 %d, min1_idx %d\n", rmin1, rmin2, idx_min);
	
//	The 2nd recursion
	for(int i = 0; i < s; i ++){
		sq = 1 - 2 * ((Q_sign >> i) & 0x01);
		R_temp = 0.75f * sign * sq * (i != idx_min ? rmin1 : rmin2);
		

		// write results to global memory
		h_element_t = dev_h_compact1[i*row+iBlkRow];
		int addr_temp = h_element_t.y * row * Zc + offsetR;
		dev_dt[addr_temp] = R_temp;
//		if(blockIdx.x == 0 && threadIdx.x == 1)printf("R_temp %d, temp_addr %d\n", R_temp, addr_temp);
	}
}

// Kernel 2: VNP processing
__global__ void
ldpc_vnp_kernel_normal(/*char * dev_llr, char * dev_dt, char * dev_const_llr,*/ int BG, int row, int col, int Zc)
{	
	int iMCW = blockIdx.y;
	int iBlkCol = blockIdx.x;
	int iBlkRow;
	int iSubCol = threadIdx.x;
	int iRow;
	int iCol;

	int shift_t, sf;
	int APP;

	h_element h_element_t;

	// update all the llr values
	iCol = iBlkCol * Zc + iSubCol;
	APP = dev_const_llr[iMCW *col*Zc + iCol];
	int offsetDt = iMCW *row*col*Zc + iBlkCol * row * Zc;
	int s = (BG==1)? h_ele_col_bg1_count[iBlkCol]:h_ele_col_bg2_count[iBlkCol];
	
	for(int i = 0; i < s; i++)
	{
		h_element_t = dev_h_compact2[i*col+iBlkCol];

		shift_t = h_element_t.value%Zc;
		iBlkRow = h_element_t.x;

		sf = iSubCol - shift_t;
		sf = (sf + Zc) % Zc;

		iRow = iBlkRow * Zc + sf;
		APP = APP + dev_dt[offsetDt + iRow];
	}
	if(APP > SCHAR_MAX)	APP = SCHAR_MAX;
	if(APP < SCHAR_MIN)	APP = SCHAR_MIN;
	// write back to device global memory
	dev_llr[iMCW *col*Zc + iCol] = APP;
}


__global__ void pack_decoded_bit(/*char *dev, unsigned char *host,*/ int col, int Zc)
{
	__shared__ unsigned char tmp[128];
	int iMCW = blockIdx.y;
	int tid = iMCW * col*Zc + blockIdx.x*128 + threadIdx.x;
	int btid = threadIdx.x;
	tmp[btid] = 0;
	
	if(dev_llr[tid] < 0){
		tmp[btid] = 1 << (7-(btid&7));
	}
	__syncthreads();
	
	if(threadIdx.x < 16){
		dev_tmp[iMCW * col*Zc + blockIdx.x*16+threadIdx.x] = 0;
		for(int i = 0; i < 8; i++){
			dev_tmp[iMCW * col*Zc + blockIdx.x*16+threadIdx.x] += tmp[threadIdx.x*8+i];
		}
	}
}

void read_BG(int BG, int *h, int row, int col)
{
	int compact_row = 30, compact_col = 19;
	if(BG==2){compact_row = 10, compact_col = 23;}
	
	h_element h_element_temp;

	// init the compact matrix
	for(int i = 0; i < compact_col; i++){
		for(int j = 0; j < row; j++){
			h_element_temp.x = 0;
			h_element_temp.y = 0;
			h_element_temp.value = -1;
			h_compact1[i*row+j] = h_element_temp; // h[i][0-11], the same column
        }
    }
	// scan the h matrix, and gengerate compact mode of h
	for(int i = 0; i < row; i++){
		int k = 0;
		for(int j = 0; j < col; j++){
			if(h[i*col+j] != -1){
				h_element_temp.x = i;
				h_element_temp.y = j;
				h_element_temp.value = h[i*col+j];
				h_compact1[k*row+i] = h_element_temp;
				k++;
            }
        }
    }
	
	// h_compact2
	// init the compact matrix
	for(int i = 0; i < compact_row; i++){
		for(int j = 0; j < col; j++){
			h_element_temp.x = 0;
			h_element_temp.y = 0;
			h_element_temp.value = -1;
			h_compact2[i*col+j] = h_element_temp;
        }
    }

	for(int j = 0; j < col; j++){
		int k=0;
		for(int i = 0; i < row; i++){
			if(h[i*col+j] != -1){
				// although h is transposed, the (x,y) is still (iBlkRow, iBlkCol)
				h_element_temp.x = i; 
				h_element_temp.y = j;
				h_element_temp.value = h[i*col+j];
				h_compact2[k*col+j] = h_element_temp;
				k++;
			}
		}
	}
	
	/*
	for(int i = 0; i < compact_col; i++){
		for(int j = 0; j < row; j++){
			printf("%3d, ", h_compact1[i*row+j].value);
		}
		printf("\n");
	}
	
	for(int i = 0; i < compact_row; i++){
		for(int j = 0; j < col; j++){
			printf("%3d,", h_compact2[i*col+j].value);
		}
		printf("\n");
	}
	*/
}

extern "C"
void init_LLR_DMA(t_nrLDPC_dec_params* p_decParams, int8_t* p_llr, int8_t* p_out){
	
	uint16_t Zc          = p_decParams->Z;
    uint8_t  BG         = p_decParams->BG;
    int block_length     = p_decParams->block_length;
	uint8_t row,col;
	if(BG == 1){
		row = 46;
		col = 68;
	}
	else{
		row = 42;
		col = 52;
	}
	unsigned char *hard_decision = (unsigned char*)p_out;
	int memorySize_llr_cuda = col * Zc * sizeof(char) * MC;
	cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_const_llr), p_llr, memorySize_llr_cuda) );
	cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_llr), p_llr, memorySize_llr_cuda) );
	hipDeviceSynchronize();
	
}

using namespace std ;

/* from here: entry points in decoder shared lib */
extern "C"
int ldpc_autoinit(void) {   // called by the library loader 
/*int devices = 0; 

  hipError_t err = hipGetDeviceCount(&devices); 
  AssertFatal(devices>0,"\nNo cuda GPU found\n\n");

    const int kb = 1024;
    const int mb = kb * kb;
    wcout << "NBody.GPU" << endl << "=========" << endl << endl;

    wcout << "CUDA version:   v" << CUDART_VERSION << endl;    
    

    wcout << "CUDA Devices: " << endl << endl;

    for(int i = 0; i < devices; ++i)
    {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        wcout << i << ": " << props.name << ": " << props.major << "." << props.minor << endl;
        wcout << "  Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
        wcout << "  Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
        wcout << "  Constant memory: " << props.totalConstMem / kb << "kb" << endl;
        wcout << "  Block registers: " << props.regsPerBlock << endl << endl;

        wcout << "  Warp size:         " << props.warpSize << endl;
        wcout << "  Threads per block: " << props.maxThreadsPerBlock << endl;
        wcout << "  Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1]  << ", " << props.maxThreadsDim[2] << " ]" << endl;
        wcout << "  Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1]  << ", " << props.maxGridSize[2] << " ]" << endl;
        wcout << endl;
    }
*/
  warmup_for_GPU();
  return 0;  
}

extern "C"
void nrLDPC_initcall(t_nrLDPC_dec_params* p_decParams, int8_t* p_llr, int8_t* p_out) {
	set_compact_BG(p_decParams->Z,p_decParams->BG);
	init_LLR_DMA(p_decParams, p_llr,  p_out);
}


extern "C"
int32_t nrLDPC_decod(t_nrLDPC_dec_params* p_decParams, int8_t* p_llr, int8_t* p_out,t_nrLDPC_procBuf* p_procBuf, t_nrLDPC_time_stats *time_decoder)
{
    uint16_t Zc          = p_decParams->Z;
    uint8_t  BG         = p_decParams->BG;
    uint8_t  numMaxIter = p_decParams->numMaxIter;
    int block_length    = p_decParams->block_length;
    e_nrLDPC_outMode outMode = p_decParams->outMode;
	hipError_t cudaStatus;
	uint8_t row,col;
	if(BG == 1){
		row = 46;
		col = 68;
	}
	else{
		row = 42;
		col = 52;
	}

//	alloc memory
	unsigned char *hard_decision = (unsigned char*)p_out;
//	gpu
	int memorySize_llr_cuda = col * Zc * sizeof(char) * MC;
	cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_const_llr), p_llr, memorySize_llr_cuda) );
	cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL(dev_llr), p_llr, memorySize_llr_cuda) );
	
// Define CUDA kernel dimension
	int blockSizeX = Zc;
	dim3 dimGridKernel1(row, MC, 1); 	// dim of the thread blocks
	dim3 dimBlockKernel1(blockSizeX, 1, 1);

    dim3 dimGridKernel2(col, MC, 1);
    dim3 dimBlockKernel2(blockSizeX, 1, 1);	
	hipDeviceSynchronize();

// lauch kernel 

	for(int ii = 0; ii < MAX_ITERATION; ii++){
		// first kernel	
		if(ii == 0){
			ldpc_cnp_kernel_1st_iter 
			<<<dimGridKernel1, dimBlockKernel1>>>
			(/*dev_llr,*/ BG, row, col, Zc);
		}else{
			ldpc_cnp_kernel
			<<<dimGridKernel1, dimBlockKernel1>>>
			(/*dev_llr,*/ BG, row, col, Zc);
		}
		// second kernel
		ldpc_vnp_kernel_normal
		<<<dimGridKernel2, dimBlockKernel2>>>
		// (dev_llr, dev_const_llr,BG, row, col, Zc);
		(BG, row, col, Zc);
	}
	
	int pack = (block_length/128)+1;
	dim3 pack_block(pack, MC, 1);
	pack_decoded_bit<<<pack_block,128>>>(/*dev_llr,*/ /*dev_tmp,*/ col, Zc);
	
	cudaCheck( hipMemcpyFromSymbol((void*)hard_decision, (const void*)dev_tmp, (block_length/8)*sizeof(unsigned char)) );
	hipDeviceSynchronize();
	

	return MAX_ITERATION;
	
}
