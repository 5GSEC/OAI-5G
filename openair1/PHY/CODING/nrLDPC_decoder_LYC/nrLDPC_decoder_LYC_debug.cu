#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "PHY/CODING/nrLDPC_decoder_LYC/nrLDPC_types.h"
#include "PHY/CODING/nrLDPC_decoder_LYC/nrLDPC_defs.h"

#define MAX_ITERATION 5
#define CW	1

#define cudaCheck(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}
// row and col element count



typedef struct{
  char x;
  char y;
  short value;
} h_element;

__device__ __constant__ char h_element_count1_bg1[46] = {
	19, 19, 19, 19, 3,  8,  9,  7, 10,  9,
	7,  8,  7,  6,  7,  7,  6,  6,  6,  6,  
	6,  6,  5,  5,  6,  5,  5,  4,  5,  5,  
	5,  5,  5,  5,  5,  5,  5,  4,  5,  5,  
	4,  5,  4,  5,  5,  4};
__device__ __constant__ char h_element_count2_bg1[68] = {
	30, 28,  7, 11,  9,  4,  8, 12,  8,  7, 
	12, 10, 12, 11, 10,  7, 10, 10, 13,  7,  
	8,  11, 12,  5,  6,  6,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,  
	1,   1,  1,  1,  1,  1,  1,  1};
__device__ __constant__ char h_element_count1_bg2[42] = { 
	8, 10,  8, 10,  4,  6,  6,  6,  4,  5, 
	5,  5,  4,  5,  5,  4,  5,  5,  4,  4,
	4,  4,  3,  4,  4,  3,  5,  3,  4,  3,
	5,  3,  4,  4,  4,  4,  4,  3,  4,  4, 
	4,  4};
__device__ __constant__ char h_element_count2_bg2[52] = {
	22, 23, 10,  5,  5, 14,  7, 13,  6,  8, 
	9,  16,  9, 12,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1,
	1,   1,  1,  1,  1,  1,  1,  1,  1,  1, 
	1,   1};

// BG
__device__ __constant__ h_element dev_h_base1_1[316]={
  { .x= 0, .y= 0, .value=307 },  { .x= 0, .y= 1, .value= 19 },  { .x= 0, .y= 2, .value= 50 },  { .x= 0, .y= 3, .value=369 },  { .x= 0, .y= 5, .value=181 },  { .x= 0, .y= 6, .value=216 },  { .x= 0, .y= 9, .value=317 },  { .x= 0, .y=10, .value=288 },  { .x= 0, .y=11, .value=109 },  { .x= 0, .y=12, .value= 17 },
  { .x= 0, .y=13, .value=357 },  { .x= 0, .y=15, .value=215 },  { .x= 0, .y=16, .value=106 },  { .x= 0, .y=18, .value=242 },  { .x= 0, .y=19, .value=180 },  { .x= 0, .y=20, .value=330 },  { .x= 0, .y=21, .value=346 },  { .x= 0, .y=22, .value=  1 },  { .x= 0, .y=23, .value=  0 },  { .x= 1, .y= 0, .value= 76 },
  { .x= 1, .y= 2, .value= 76 },  { .x= 1, .y= 3, .value= 73 },  { .x= 1, .y= 4, .value=288 },  { .x= 1, .y= 5, .value=144 },  { .x= 1, .y= 7, .value=331 },  { .x= 1, .y= 8, .value=331 },  { .x= 1, .y= 9, .value=178 },  { .x= 1, .y=11, .value=295 },  { .x= 1, .y=12, .value=342 },  { .x= 1, .y=14, .value=217 },
  { .x= 1, .y=15, .value= 99 },  { .x= 1, .y=16, .value=354 },  { .x= 1, .y=17, .value=114 },  { .x= 1, .y=19, .value=331 },  { .x= 1, .y=21, .value=112 },  { .x= 1, .y=22, .value=  0 },  { .x= 1, .y=23, .value=  0 },  { .x= 1, .y=24, .value=  0 },  { .x= 2, .y= 0, .value=205 },  { .x= 2, .y= 1, .value=250 },
  { .x= 2, .y= 2, .value=328 },  { .x= 2, .y= 4, .value=332 },  { .x= 2, .y= 5, .value=256 },  { .x= 2, .y= 6, .value=161 },  { .x= 2, .y= 7, .value=267 },  { .x= 2, .y= 8, .value=160 },  { .x= 2, .y= 9, .value= 63 },  { .x= 2, .y=10, .value=129 },  { .x= 2, .y=13, .value=200 },  { .x= 2, .y=14, .value= 88 },
  { .x= 2, .y=15, .value= 53 },  { .x= 2, .y=17, .value=131 },  { .x= 2, .y=18, .value=240 },  { .x= 2, .y=19, .value=205 },  { .x= 2, .y=20, .value= 13 },  { .x= 2, .y=24, .value=  0 },  { .x= 2, .y=25, .value=  0 },  { .x= 3, .y= 0, .value=276 },  { .x= 3, .y= 1, .value= 87 },  { .x= 3, .y= 3, .value=  0 },
  { .x= 3, .y= 4, .value=275 },  { .x= 3, .y= 6, .value=199 },  { .x= 3, .y= 7, .value=153 },  { .x= 3, .y= 8, .value= 56 },  { .x= 3, .y=10, .value=132 },  { .x= 3, .y=11, .value=305 },  { .x= 3, .y=12, .value=231 },  { .x= 3, .y=13, .value=341 },  { .x= 3, .y=14, .value=212 },  { .x= 3, .y=16, .value=304 },
  { .x= 3, .y=17, .value=300 },  { .x= 3, .y=18, .value=271 },  { .x= 3, .y=20, .value= 39 },  { .x= 3, .y=21, .value=357 },  { .x= 3, .y=22, .value=  1 },  { .x= 3, .y=25, .value=  0 },  { .x= 4, .y= 0, .value=332 },  { .x= 4, .y= 1, .value=181 },  { .x= 4, .y=26, .value=  0 },  { .x= 5, .y= 0, .value=195 },
  { .x= 5, .y= 1, .value= 14 },  { .x= 5, .y= 3, .value=115 },  { .x= 5, .y=12, .value=166 },  { .x= 5, .y=16, .value=241 },  { .x= 5, .y=21, .value= 51 },  { .x= 5, .y=22, .value=157 },  { .x= 5, .y=27, .value=  0 },  { .x= 6, .y= 0, .value=278 },  { .x= 6, .y= 6, .value=257 },  { .x= 6, .y=10, .value=  1 },
  { .x= 6, .y=11, .value=351 },  { .x= 6, .y=13, .value= 92 },  { .x= 6, .y=17, .value=253 },  { .x= 6, .y=18, .value= 18 },  { .x= 6, .y=20, .value=225 },  { .x= 6, .y=28, .value=  0 },  { .x= 7, .y= 0, .value=  9 },  { .x= 7, .y= 1, .value= 62 },  { .x= 7, .y= 4, .value=316 },  { .x= 7, .y= 7, .value=333 },
  { .x= 7, .y= 8, .value=290 },  { .x= 7, .y=14, .value=114 },  { .x= 7, .y=29, .value=  0 },  { .x= 8, .y= 0, .value=307 },  { .x= 8, .y= 1, .value=179 },  { .x= 8, .y= 3, .value=165 },  { .x= 8, .y=12, .value= 18 },  { .x= 8, .y=16, .value= 39 },  { .x= 8, .y=19, .value=224 },  { .x= 8, .y=21, .value=368 },
  { .x= 8, .y=22, .value= 67 },  { .x= 8, .y=24, .value=170 },  { .x= 8, .y=30, .value=  0 },  { .x= 9, .y= 0, .value=366 },  { .x= 9, .y= 1, .value=232 },  { .x= 9, .y=10, .value=321 },  { .x= 9, .y=11, .value=133 },  { .x= 9, .y=13, .value= 57 },  { .x= 9, .y=17, .value=303 },  { .x= 9, .y=18, .value= 63 },
  { .x= 9, .y=20, .value= 82 },  { .x= 9, .y=31, .value=  0 },  { .x=10, .y= 1, .value=101 },  { .x=10, .y= 2, .value=339 },  { .x=10, .y= 4, .value=274 },  { .x=10, .y= 7, .value=111 },  { .x=10, .y= 8, .value=383 },  { .x=10, .y=14, .value=354 },  { .x=10, .y=32, .value=  0 },  { .x=11, .y= 0, .value= 48 },
  { .x=11, .y= 1, .value=102 },  { .x=11, .y=12, .value=  8 },  { .x=11, .y=16, .value= 47 },  { .x=11, .y=21, .value=188 },  { .x=11, .y=22, .value=334 },  { .x=11, .y=23, .value=115 },  { .x=11, .y=33, .value=  0 },  { .x=12, .y= 0, .value= 77 },  { .x=12, .y= 1, .value=186 },  { .x=12, .y=10, .value=174 },
  { .x=12, .y=11, .value=232 },  { .x=12, .y=13, .value= 50 },  { .x=12, .y=18, .value= 74 },  { .x=12, .y=34, .value=  0 },  { .x=13, .y= 0, .value=313 },  { .x=13, .y= 3, .value=177 },  { .x=13, .y= 7, .value=266 },  { .x=13, .y=20, .value=115 },  { .x=13, .y=23, .value=370 },  { .x=13, .y=35, .value=  0 },
  { .x=14, .y= 0, .value=142 },  { .x=14, .y=12, .value=248 },  { .x=14, .y=15, .value=137 },  { .x=14, .y=16, .value= 89 },  { .x=14, .y=17, .value=347 },  { .x=14, .y=21, .value= 12 },  { .x=14, .y=36, .value=  0 },  { .x=15, .y= 0, .value=241 },  { .x=15, .y= 1, .value=  2 },  { .x=15, .y=10, .value=210 },
  { .x=15, .y=13, .value=318 },  { .x=15, .y=18, .value= 55 },  { .x=15, .y=25, .value=269 },  { .x=15, .y=37, .value=  0 },  { .x=16, .y= 1, .value= 13 },  { .x=16, .y= 3, .value=338 },  { .x=16, .y=11, .value= 57 },  { .x=16, .y=20, .value=289 },  { .x=16, .y=22, .value= 57 },  { .x=16, .y=38, .value=  0 },
  { .x=17, .y= 0, .value=260 },  { .x=17, .y=14, .value=303 },  { .x=17, .y=16, .value= 81 },  { .x=17, .y=17, .value=358 },  { .x=17, .y=21, .value=375 },  { .x=17, .y=39, .value=  0 },  { .x=18, .y= 1, .value=130 },  { .x=18, .y=12, .value=163 },  { .x=18, .y=13, .value=280 },  { .x=18, .y=18, .value=132 },
  { .x=18, .y=19, .value=  4 },  { .x=18, .y=40, .value=  0 },  { .x=19, .y= 0, .value=145 },  { .x=19, .y= 1, .value=213 },  { .x=19, .y= 7, .value=344 },  { .x=19, .y= 8, .value=242 },  { .x=19, .y=10, .value=197 },  { .x=19, .y=41, .value=  0 },  { .x=20, .y= 0, .value=187 },  { .x=20, .y= 3, .value=206 },
  { .x=20, .y= 9, .value=264 },  { .x=20, .y=11, .value=341 },  { .x=20, .y=22, .value= 59 },  { .x=20, .y=42, .value=  0 },  { .x=21, .y= 1, .value=205 },  { .x=21, .y= 5, .value=102 },  { .x=21, .y=16, .value=328 },  { .x=21, .y=20, .value=213 },  { .x=21, .y=21, .value= 97 },  { .x=21, .y=43, .value=  0 },
  { .x=22, .y= 0, .value= 30 },  { .x=22, .y=12, .value= 11 },  { .x=22, .y=13, .value=233 },  { .x=22, .y=17, .value= 22 },  { .x=22, .y=44, .value=  0 },  { .x=23, .y= 1, .value= 24 },  { .x=23, .y= 2, .value= 89 },  { .x=23, .y=10, .value= 61 },  { .x=23, .y=18, .value= 27 },  { .x=23, .y=45, .value=  0 },
  { .x=24, .y= 0, .value=298 },  { .x=24, .y= 3, .value=158 },  { .x=24, .y= 4, .value=235 },  { .x=24, .y=11, .value=339 },  { .x=24, .y=22, .value=234 },  { .x=24, .y=46, .value=  0 },  { .x=25, .y= 1, .value= 72 },  { .x=25, .y= 6, .value= 17 },  { .x=25, .y= 7, .value=383 },  { .x=25, .y=14, .value=312 },
  { .x=25, .y=47, .value=  0 },  { .x=26, .y= 0, .value= 71 },  { .x=26, .y= 2, .value= 81 },  { .x=26, .y= 4, .value= 76 },  { .x=26, .y=15, .value=136 },  { .x=26, .y=48, .value=  0 },  { .x=27, .y= 1, .value=194 },  { .x=27, .y= 6, .value=194 },  { .x=27, .y= 8, .value=101 },  { .x=27, .y=49, .value=  0 },
  { .x=28, .y= 0, .value=222 },  { .x=28, .y= 4, .value= 19 },  { .x=28, .y=19, .value=244 },  { .x=28, .y=21, .value=274 },  { .x=28, .y=50, .value=  0 },  { .x=29, .y= 1, .value=252 },  { .x=29, .y=14, .value=  5 },  { .x=29, .y=18, .value=147 },  { .x=29, .y=25, .value= 78 },  { .x=29, .y=51, .value=  0 },
  { .x=30, .y= 0, .value=159 },  { .x=30, .y=10, .value=229 },  { .x=30, .y=13, .value=260 },  { .x=30, .y=24, .value= 90 },  { .x=30, .y=52, .value=  0 },  { .x=31, .y= 1, .value=100 },  { .x=31, .y= 7, .value=215 },  { .x=31, .y=22, .value=258 },  { .x=31, .y=25, .value=256 },  { .x=31, .y=53, .value=  0 },
  { .x=32, .y= 0, .value=102 },  { .x=32, .y=12, .value=201 },  { .x=32, .y=14, .value=175 },  { .x=32, .y=24, .value=287 },  { .x=32, .y=54, .value=  0 },  { .x=33, .y= 1, .value=323 },  { .x=33, .y= 2, .value=  8 },  { .x=33, .y=11, .value=361 },  { .x=33, .y=21, .value=105 },  { .x=33, .y=55, .value=  0 },
  { .x=34, .y= 0, .value=230 },  { .x=34, .y= 7, .value=148 },  { .x=34, .y=15, .value=202 },  { .x=34, .y=17, .value=312 },  { .x=34, .y=56, .value=  0 },  { .x=35, .y= 1, .value=320 },  { .x=35, .y= 6, .value=335 },  { .x=35, .y=12, .value=  2 },  { .x=35, .y=22, .value=266 },  { .x=35, .y=57, .value=  0 },
  { .x=36, .y= 0, .value=210 },  { .x=36, .y=14, .value=313 },  { .x=36, .y=15, .value=297 },  { .x=36, .y=18, .value= 21 },  { .x=36, .y=58, .value=  0 },  { .x=37, .y= 1, .value=269 },  { .x=37, .y=13, .value= 82 },  { .x=37, .y=23, .value=115 },  { .x=37, .y=59, .value=  0 },  { .x=38, .y= 0, .value=185 },
  { .x=38, .y= 9, .value=177 },  { .x=38, .y=10, .value=289 },  { .x=38, .y=12, .value=214 },  { .x=38, .y=60, .value=  0 },  { .x=39, .y= 1, .value=258 },  { .x=39, .y= 3, .value= 93 },  { .x=39, .y= 7, .value=346 },  { .x=39, .y=19, .value=297 },  { .x=39, .y=61, .value=  0 },  { .x=40, .y= 0, .value=175 },
  { .x=40, .y= 8, .value= 37 },  { .x=40, .y=17, .value=312 },  { .x=40, .y=62, .value=  0 },  { .x=41, .y= 1, .value= 52 },  { .x=41, .y= 3, .value=314 },  { .x=41, .y= 9, .value=139 },  { .x=41, .y=18, .value=288 },  { .x=41, .y=63, .value=  0 },  { .x=42, .y= 0, .value=113 },  { .x=42, .y= 4, .value= 14 },
  { .x=42, .y=24, .value=218 },  { .x=42, .y=64, .value=  0 },  { .x=43, .y= 1, .value=113 },  { .x=43, .y=16, .value=132 },  { .x=43, .y=18, .value=114 },  { .x=43, .y=25, .value=168 },  { .x=43, .y=65, .value=  0 },  { .x=44, .y= 0, .value= 80 },  { .x=44, .y= 7, .value= 78 },  { .x=44, .y= 9, .value=163 },
  { .x=44, .y=22, .value=274 },  { .x=44, .y=66, .value=  0 },  { .x=45, .y= 1, .value=135 },  { .x=45, .y= 6, .value=149 },  { .x=45, .y=10, .value= 15 },  { .x=45, .y=67, .value=  0 } };
__device__ __constant__ h_element* dev_h_compact1[46]={
  &(dev_h_base1_1[  0]),  &(dev_h_base1_1[ 19]),  &(dev_h_base1_1[ 38]),  &(dev_h_base1_1[ 57]),  &(dev_h_base1_1[ 76]),  &(dev_h_base1_1[ 79]),  &(dev_h_base1_1[ 87]),  &(dev_h_base1_1[ 96]),  &(dev_h_base1_1[103]),  &(dev_h_base1_1[113]),
  &(dev_h_base1_1[122]),  &(dev_h_base1_1[129]),  &(dev_h_base1_1[137]),  &(dev_h_base1_1[144]),  &(dev_h_base1_1[150]),  &(dev_h_base1_1[157]),  &(dev_h_base1_1[164]),  &(dev_h_base1_1[170]),  &(dev_h_base1_1[176]),  &(dev_h_base1_1[182]),
  &(dev_h_base1_1[188]),  &(dev_h_base1_1[194]),  &(dev_h_base1_1[200]),  &(dev_h_base1_1[205]),  &(dev_h_base1_1[210]),  &(dev_h_base1_1[216]),  &(dev_h_base1_1[221]),  &(dev_h_base1_1[226]),  &(dev_h_base1_1[230]),  &(dev_h_base1_1[235]),
  &(dev_h_base1_1[240]),  &(dev_h_base1_1[245]),  &(dev_h_base1_1[250]),  &(dev_h_base1_1[255]),  &(dev_h_base1_1[260]),  &(dev_h_base1_1[265]),  &(dev_h_base1_1[270]),  &(dev_h_base1_1[275]),  &(dev_h_base1_1[279]),  &(dev_h_base1_1[284]),
  &(dev_h_base1_1[289]),  &(dev_h_base1_1[293]),  &(dev_h_base1_1[298]),  &(dev_h_base1_1[302]),  &(dev_h_base1_1[307]),  &(dev_h_base1_1[312]) };
__device__ __constant__ h_element dev_h_base2_1[316]={
  { .x= 0, .y= 0, .value=307 },  { .x= 1, .y= 0, .value= 76 },  { .x= 2, .y= 0, .value=205 },  { .x= 3, .y= 0, .value=276 },  { .x= 4, .y= 0, .value=332 },  { .x= 5, .y= 0, .value=195 },  { .x= 6, .y= 0, .value=278 },  { .x= 7, .y= 0, .value=  9 },  { .x= 8, .y= 0, .value=307 },  { .x= 9, .y= 0, .value=366 },
  { .x=11, .y= 0, .value= 48 },  { .x=12, .y= 0, .value= 77 },  { .x=13, .y= 0, .value=313 },  { .x=14, .y= 0, .value=142 },  { .x=15, .y= 0, .value=241 },  { .x=17, .y= 0, .value=260 },  { .x=19, .y= 0, .value=145 },  { .x=20, .y= 0, .value=187 },  { .x=22, .y= 0, .value= 30 },  { .x=24, .y= 0, .value=298 },
  { .x=26, .y= 0, .value= 71 },  { .x=28, .y= 0, .value=222 },  { .x=30, .y= 0, .value=159 },  { .x=32, .y= 0, .value=102 },  { .x=34, .y= 0, .value=230 },  { .x=36, .y= 0, .value=210 },  { .x=38, .y= 0, .value=185 },  { .x=40, .y= 0, .value=175 },  { .x=42, .y= 0, .value=113 },  { .x=44, .y= 0, .value= 80 },
  { .x= 0, .y= 1, .value= 19 },  { .x= 2, .y= 1, .value=250 },  { .x= 3, .y= 1, .value= 87 },  { .x= 4, .y= 1, .value=181 },  { .x= 5, .y= 1, .value= 14 },  { .x= 7, .y= 1, .value= 62 },  { .x= 8, .y= 1, .value=179 },  { .x= 9, .y= 1, .value=232 },  { .x=10, .y= 1, .value=101 },  { .x=11, .y= 1, .value=102 },
  { .x=12, .y= 1, .value=186 },  { .x=15, .y= 1, .value=  2 },  { .x=16, .y= 1, .value= 13 },  { .x=18, .y= 1, .value=130 },  { .x=19, .y= 1, .value=213 },  { .x=21, .y= 1, .value=205 },  { .x=23, .y= 1, .value= 24 },  { .x=25, .y= 1, .value= 72 },  { .x=27, .y= 1, .value=194 },  { .x=29, .y= 1, .value=252 },
  { .x=31, .y= 1, .value=100 },  { .x=33, .y= 1, .value=323 },  { .x=35, .y= 1, .value=320 },  { .x=37, .y= 1, .value=269 },  { .x=39, .y= 1, .value=258 },  { .x=41, .y= 1, .value= 52 },  { .x=43, .y= 1, .value=113 },  { .x=45, .y= 1, .value=135 },  { .x= 0, .y= 2, .value= 50 },  { .x= 1, .y= 2, .value= 76 },
  { .x= 2, .y= 2, .value=328 },  { .x=10, .y= 2, .value=339 },  { .x=23, .y= 2, .value= 89 },  { .x=26, .y= 2, .value= 81 },  { .x=33, .y= 2, .value=  8 },  { .x= 0, .y= 3, .value=369 },  { .x= 1, .y= 3, .value= 73 },  { .x= 3, .y= 3, .value=  0 },  { .x= 5, .y= 3, .value=115 },  { .x= 8, .y= 3, .value=165 },
  { .x=13, .y= 3, .value=177 },  { .x=16, .y= 3, .value=338 },  { .x=20, .y= 3, .value=206 },  { .x=24, .y= 3, .value=158 },  { .x=39, .y= 3, .value= 93 },  { .x=41, .y= 3, .value=314 },  { .x= 1, .y= 4, .value=288 },  { .x= 2, .y= 4, .value=332 },  { .x= 3, .y= 4, .value=275 },  { .x= 7, .y= 4, .value=316 },
  { .x=10, .y= 4, .value=274 },  { .x=24, .y= 4, .value=235 },  { .x=26, .y= 4, .value= 76 },  { .x=28, .y= 4, .value= 19 },  { .x=42, .y= 4, .value= 14 },  { .x= 0, .y= 5, .value=181 },  { .x= 1, .y= 5, .value=144 },  { .x= 2, .y= 5, .value=256 },  { .x=21, .y= 5, .value=102 },  { .x= 0, .y= 6, .value=216 },
  { .x= 2, .y= 6, .value=161 },  { .x= 3, .y= 6, .value=199 },  { .x= 6, .y= 6, .value=257 },  { .x=25, .y= 6, .value= 17 },  { .x=27, .y= 6, .value=194 },  { .x=35, .y= 6, .value=335 },  { .x=45, .y= 6, .value=149 },  { .x= 1, .y= 7, .value=331 },  { .x= 2, .y= 7, .value=267 },  { .x= 3, .y= 7, .value=153 },
  { .x= 7, .y= 7, .value=333 },  { .x=10, .y= 7, .value=111 },  { .x=13, .y= 7, .value=266 },  { .x=19, .y= 7, .value=344 },  { .x=25, .y= 7, .value=383 },  { .x=31, .y= 7, .value=215 },  { .x=34, .y= 7, .value=148 },  { .x=39, .y= 7, .value=346 },  { .x=44, .y= 7, .value= 78 },  { .x= 1, .y= 8, .value=331 },
  { .x= 2, .y= 8, .value=160 },  { .x= 3, .y= 8, .value= 56 },  { .x= 7, .y= 8, .value=290 },  { .x=10, .y= 8, .value=383 },  { .x=19, .y= 8, .value=242 },  { .x=27, .y= 8, .value=101 },  { .x=40, .y= 8, .value= 37 },  { .x= 0, .y= 9, .value=317 },  { .x= 1, .y= 9, .value=178 },  { .x= 2, .y= 9, .value= 63 },
  { .x=20, .y= 9, .value=264 },  { .x=38, .y= 9, .value=177 },  { .x=41, .y= 9, .value=139 },  { .x=44, .y= 9, .value=163 },  { .x= 0, .y=10, .value=288 },  { .x= 2, .y=10, .value=129 },  { .x= 3, .y=10, .value=132 },  { .x= 6, .y=10, .value=  1 },  { .x= 9, .y=10, .value=321 },  { .x=12, .y=10, .value=174 },
  { .x=15, .y=10, .value=210 },  { .x=19, .y=10, .value=197 },  { .x=23, .y=10, .value= 61 },  { .x=30, .y=10, .value=229 },  { .x=38, .y=10, .value=289 },  { .x=45, .y=10, .value= 15 },  { .x= 0, .y=11, .value=109 },  { .x= 1, .y=11, .value=295 },  { .x= 3, .y=11, .value=305 },  { .x= 6, .y=11, .value=351 },
  { .x= 9, .y=11, .value=133 },  { .x=12, .y=11, .value=232 },  { .x=16, .y=11, .value= 57 },  { .x=20, .y=11, .value=341 },  { .x=24, .y=11, .value=339 },  { .x=33, .y=11, .value=361 },  { .x= 0, .y=12, .value= 17 },  { .x= 1, .y=12, .value=342 },  { .x= 3, .y=12, .value=231 },  { .x= 5, .y=12, .value=166 },
  { .x= 8, .y=12, .value= 18 },  { .x=11, .y=12, .value=  8 },  { .x=14, .y=12, .value=248 },  { .x=18, .y=12, .value=163 },  { .x=22, .y=12, .value= 11 },  { .x=32, .y=12, .value=201 },  { .x=35, .y=12, .value=  2 },  { .x=38, .y=12, .value=214 },  { .x= 0, .y=13, .value=357 },  { .x= 2, .y=13, .value=200 },
  { .x= 3, .y=13, .value=341 },  { .x= 6, .y=13, .value= 92 },  { .x= 9, .y=13, .value= 57 },  { .x=12, .y=13, .value= 50 },  { .x=15, .y=13, .value=318 },  { .x=18, .y=13, .value=280 },  { .x=22, .y=13, .value=233 },  { .x=30, .y=13, .value=260 },  { .x=37, .y=13, .value= 82 },  { .x= 1, .y=14, .value=217 },
  { .x= 2, .y=14, .value= 88 },  { .x= 3, .y=14, .value=212 },  { .x= 7, .y=14, .value=114 },  { .x=10, .y=14, .value=354 },  { .x=17, .y=14, .value=303 },  { .x=25, .y=14, .value=312 },  { .x=29, .y=14, .value=  5 },  { .x=32, .y=14, .value=175 },  { .x=36, .y=14, .value=313 },  { .x= 0, .y=15, .value=215 },
  { .x= 1, .y=15, .value= 99 },  { .x= 2, .y=15, .value= 53 },  { .x=14, .y=15, .value=137 },  { .x=26, .y=15, .value=136 },  { .x=34, .y=15, .value=202 },  { .x=36, .y=15, .value=297 },  { .x= 0, .y=16, .value=106 },  { .x= 1, .y=16, .value=354 },  { .x= 3, .y=16, .value=304 },  { .x= 5, .y=16, .value=241 },
  { .x= 8, .y=16, .value= 39 },  { .x=11, .y=16, .value= 47 },  { .x=14, .y=16, .value= 89 },  { .x=17, .y=16, .value= 81 },  { .x=21, .y=16, .value=328 },  { .x=43, .y=16, .value=132 },  { .x= 1, .y=17, .value=114 },  { .x= 2, .y=17, .value=131 },  { .x= 3, .y=17, .value=300 },  { .x= 6, .y=17, .value=253 },
  { .x= 9, .y=17, .value=303 },  { .x=14, .y=17, .value=347 },  { .x=17, .y=17, .value=358 },  { .x=22, .y=17, .value= 22 },  { .x=34, .y=17, .value=312 },  { .x=40, .y=17, .value=312 },  { .x= 0, .y=18, .value=242 },  { .x= 2, .y=18, .value=240 },  { .x= 3, .y=18, .value=271 },  { .x= 6, .y=18, .value= 18 },
  { .x= 9, .y=18, .value= 63 },  { .x=12, .y=18, .value= 74 },  { .x=15, .y=18, .value= 55 },  { .x=18, .y=18, .value=132 },  { .x=23, .y=18, .value= 27 },  { .x=29, .y=18, .value=147 },  { .x=36, .y=18, .value= 21 },  { .x=41, .y=18, .value=288 },  { .x=43, .y=18, .value=114 },  { .x= 0, .y=19, .value=180 },
  { .x= 1, .y=19, .value=331 },  { .x= 2, .y=19, .value=205 },  { .x= 8, .y=19, .value=224 },  { .x=18, .y=19, .value=  4 },  { .x=28, .y=19, .value=244 },  { .x=39, .y=19, .value=297 },  { .x= 0, .y=20, .value=330 },  { .x= 2, .y=20, .value= 13 },  { .x= 3, .y=20, .value= 39 },  { .x= 6, .y=20, .value=225 },
  { .x= 9, .y=20, .value= 82 },  { .x=13, .y=20, .value=115 },  { .x=16, .y=20, .value=289 },  { .x=21, .y=20, .value=213 },  { .x= 0, .y=21, .value=346 },  { .x= 1, .y=21, .value=112 },  { .x= 3, .y=21, .value=357 },  { .x= 5, .y=21, .value= 51 },  { .x= 8, .y=21, .value=368 },  { .x=11, .y=21, .value=188 },
  { .x=14, .y=21, .value= 12 },  { .x=17, .y=21, .value=375 },  { .x=21, .y=21, .value= 97 },  { .x=28, .y=21, .value=274 },  { .x=33, .y=21, .value=105 },  { .x= 0, .y=22, .value=  1 },  { .x= 1, .y=22, .value=  0 },  { .x= 3, .y=22, .value=  1 },  { .x= 5, .y=22, .value=157 },  { .x= 8, .y=22, .value= 67 },
  { .x=11, .y=22, .value=334 },  { .x=16, .y=22, .value= 57 },  { .x=20, .y=22, .value= 59 },  { .x=24, .y=22, .value=234 },  { .x=31, .y=22, .value=258 },  { .x=35, .y=22, .value=266 },  { .x=44, .y=22, .value=274 },  { .x= 0, .y=23, .value=  0 },  { .x= 1, .y=23, .value=  0 },  { .x=11, .y=23, .value=115 },
  { .x=13, .y=23, .value=370 },  { .x=37, .y=23, .value=115 },  { .x= 1, .y=24, .value=  0 },  { .x= 2, .y=24, .value=  0 },  { .x= 8, .y=24, .value=170 },  { .x=30, .y=24, .value= 90 },  { .x=32, .y=24, .value=287 },  { .x=42, .y=24, .value=218 },  { .x= 2, .y=25, .value=  0 },  { .x= 3, .y=25, .value=  0 },
  { .x=15, .y=25, .value=269 },  { .x=29, .y=25, .value= 78 },  { .x=31, .y=25, .value=256 },  { .x=43, .y=25, .value=168 },  { .x= 4, .y=26, .value=  0 },  { .x= 5, .y=27, .value=  0 },  { .x= 6, .y=28, .value=  0 },  { .x= 7, .y=29, .value=  0 },  { .x= 8, .y=30, .value=  0 },  { .x= 9, .y=31, .value=  0 },
  { .x=10, .y=32, .value=  0 },  { .x=11, .y=33, .value=  0 },  { .x=12, .y=34, .value=  0 },  { .x=13, .y=35, .value=  0 },  { .x=14, .y=36, .value=  0 },  { .x=15, .y=37, .value=  0 },  { .x=16, .y=38, .value=  0 },  { .x=17, .y=39, .value=  0 },  { .x=18, .y=40, .value=  0 },  { .x=19, .y=41, .value=  0 },
  { .x=20, .y=42, .value=  0 },  { .x=21, .y=43, .value=  0 },  { .x=22, .y=44, .value=  0 },  { .x=23, .y=45, .value=  0 },  { .x=24, .y=46, .value=  0 },  { .x=25, .y=47, .value=  0 },  { .x=26, .y=48, .value=  0 },  { .x=27, .y=49, .value=  0 },  { .x=28, .y=50, .value=  0 },  { .x=29, .y=51, .value=  0 },
  { .x=30, .y=52, .value=  0 },  { .x=31, .y=53, .value=  0 },  { .x=32, .y=54, .value=  0 },  { .x=33, .y=55, .value=  0 },  { .x=34, .y=56, .value=  0 },  { .x=35, .y=57, .value=  0 },  { .x=36, .y=58, .value=  0 },  { .x=37, .y=59, .value=  0 },  { .x=38, .y=60, .value=  0 },  { .x=39, .y=61, .value=  0 },
  { .x=40, .y=62, .value=  0 },  { .x=41, .y=63, .value=  0 },  { .x=42, .y=64, .value=  0 },  { .x=43, .y=65, .value=  0 },  { .x=44, .y=66, .value=  0 },  { .x=45, .y=67, .value=  0 } };
__device__ __constant__ h_element* dev_h_compact2[68]={
  &(dev_h_base2_1[  0]),  &(dev_h_base2_1[ 30]),  &(dev_h_base2_1[ 58]),  &(dev_h_base2_1[ 65]),  &(dev_h_base2_1[ 76]),  &(dev_h_base2_1[ 85]),  &(dev_h_base2_1[ 89]),  &(dev_h_base2_1[ 97]),  &(dev_h_base2_1[109]),  &(dev_h_base2_1[117]),
  &(dev_h_base2_1[124]),  &(dev_h_base2_1[136]),  &(dev_h_base2_1[146]),  &(dev_h_base2_1[158]),  &(dev_h_base2_1[169]),  &(dev_h_base2_1[179]),  &(dev_h_base2_1[186]),  &(dev_h_base2_1[196]),  &(dev_h_base2_1[206]),  &(dev_h_base2_1[219]),
  &(dev_h_base2_1[226]),  &(dev_h_base2_1[234]),  &(dev_h_base2_1[245]),  &(dev_h_base2_1[257]),  &(dev_h_base2_1[262]),  &(dev_h_base2_1[268]),  &(dev_h_base2_1[274]),  &(dev_h_base2_1[275]),  &(dev_h_base2_1[276]),  &(dev_h_base2_1[277]),
  &(dev_h_base2_1[278]),  &(dev_h_base2_1[279]),  &(dev_h_base2_1[280]),  &(dev_h_base2_1[281]),  &(dev_h_base2_1[282]),  &(dev_h_base2_1[283]),  &(dev_h_base2_1[284]),  &(dev_h_base2_1[285]),  &(dev_h_base2_1[286]),  &(dev_h_base2_1[287]),
  &(dev_h_base2_1[288]),  &(dev_h_base2_1[289]),  &(dev_h_base2_1[290]),  &(dev_h_base2_1[291]),  &(dev_h_base2_1[292]),  &(dev_h_base2_1[293]),  &(dev_h_base2_1[294]),  &(dev_h_base2_1[295]),  &(dev_h_base2_1[296]),  &(dev_h_base2_1[297]),
  &(dev_h_base2_1[298]),  &(dev_h_base2_1[299]),  &(dev_h_base2_1[300]),  &(dev_h_base2_1[301]),  &(dev_h_base2_1[302]),  &(dev_h_base2_1[303]),  &(dev_h_base2_1[304]),  &(dev_h_base2_1[305]),  &(dev_h_base2_1[306]),  &(dev_h_base2_1[307]),
  &(dev_h_base2_1[308]),  &(dev_h_base2_1[309]),  &(dev_h_base2_1[310]),  &(dev_h_base2_1[311]),  &(dev_h_base2_1[312]),  &(dev_h_base2_1[313]),  &(dev_h_base2_1[314]),  &(dev_h_base2_1[315]) };




__device__ void cnp_1st(int row, char* dev_llr, char* dev_buf, int blk, int thd)
{
	int subrow = threadIdx.x, subcol = threadIdx.x, Zc = blockDim.x;
	
	for(int i = 0; i < row; i++)
	{
		int cnt = h_element_count1_bg1[i];
		int min1 = INT32_MAX, min2 = INT32_MAX;
		int sign = 0, tsign = 1, idx = 0;
		int irow = subrow + Zc*i;
		
		for(int j = 0; j < cnt; j++)
		{
			h_element tmp = dev_h_compact1[i][j];
			int icol = tmp.y * Zc + (subcol+tmp.value)%Zc;
			
			int Q = dev_llr[icol];
			int Q_abs = (Q>0)? Q:-Q;
			char sq = (Q<0);
			tsign = tsign*(1-sq*2);
			sign |= (sq << j);
			
			if(threadIdx.x == thd && i == blk){
				printf("dev_llr[%d](Q): %d, total_sign: %d, sign: %d\n", icol, dev_llr[icol], tsign, sign);
			}
			
			
			if(Q_abs < min1){
				min2 = min1;
				min1 = Q_abs;
				idx = j;
			}else if(Q_abs < min2){
				min2 = Q_abs;
			}
		}
		
		if(threadIdx.x == thd && i == blk){
			printf("sign: %d, idx: %d, min1: %d, min2: %d\n", sign, idx, min1, min2);
		}
		
		
		for(int j = 0; j < cnt; j++)
		{
			char sq = 1 - 2 * ((sign >> j) & 0x01);
			int temp = tsign * sq * ((j != idx)? min1 : min2);
			h_element tmp = dev_h_compact1[i][j];
			int addr = irow + tmp.y * row * Zc;
			dev_buf[addr] = temp;
			
			if(threadIdx.x == thd && i == blk){
				printf("temp: %d, addr: %d\n", temp, addr);
			}
		}
	}
	__syncthreads();
}

__device__ void bnp(int row, int col, char* dev_llr, char* dev_const_llr, char* dev_buf, int blk, int thd)
{
	int subrow = threadIdx.x, subcol = threadIdx.x, Zc = blockDim.x;
	for(int i = 0; i < col; i++)
	{
		int cnt = h_element_count2_bg1[i];
		int icol = i * Zc + subcol;
		int sum = dev_const_llr[icol];
		
		for(int j = 0; j < cnt; j++)
		{
			h_element tmp = dev_h_compact2[i][j];
			int irow = tmp.x * Zc + (subrow + Zc - tmp.value)%Zc;
			int addr = i * row * Zc + irow;
			sum = sum + dev_buf[addr];
			
			if(threadIdx.x == thd && i == blk){
				printf("dev_buf[%d]: %d, sum: %d, x: %d, value: %d, irow: %d\n", addr, dev_buf[addr], sum, tmp.x, tmp.value, irow);
			}
			
		}
		if(sum > SCHAR_MAX)	sum = SCHAR_MAX;
		if(sum < SCHAR_MIN)	sum = SCHAR_MIN;
		dev_llr[icol] = sum;
	}
	__syncthreads();
}


__device__ void cnp(int row, char* dev_llr, char* dev_buf, int blk, int thd)
{
	int subrow = threadIdx.x, subcol = threadIdx.x, Zc = blockDim.x;
	
	for(int i = 0; i < row; i++)
	{
		int cnt = h_element_count1_bg1[i];
		int min1 = INT32_MAX, min2 = INT32_MAX;
		int sign = 0, tsign = 1, idx = 0;
		int irow = subrow + Zc*i;
		
		for(int j = 0; j < cnt; j++)
		{
			h_element tmp = dev_h_compact1[i][j];
			int icol = tmp.y * Zc + (subcol+tmp.value)%Zc;
			
			int Q_last = dev_buf[tmp.y * row * Zc + irow];
			
			int Q = dev_llr[icol] - Q_last;
			int Q_abs = (Q>0)? Q:-Q;
			char sq = (Q<0);
			tsign = tsign*(1-sq*2);
			sign |= (sq << j);
			
			if(threadIdx.x == thd && i == blk){
				printf("dev_llr[%d](Q): %d, total_sign: %d, sign: %d, last Q: %d, last_pos: %d\n", icol, dev_llr[icol], tsign, sign, Q_last, tmp.y * row * Zc + irow);
			}
			
			
			if(Q_abs < min1){
				min2 = min1;
				min1 = Q_abs;
				idx = j;
			}else if(Q_abs < min2){
				min2 = Q_abs;
			}
		}
		
		if(threadIdx.x == thd && i == blk){
			printf("sign: %d, idx: %d, min1: %d, min2: %d\n", sign, idx, min1, min2);
		}
		
		
		for(int j = 0; j < cnt; j++)
		{
			char sq = 1 - 2 * ((sign >> j) & 0x01);
			int temp = tsign * sq * ((j != idx)? min1 : min2);
			h_element tmp = dev_h_compact1[i][j];
			int addr = irow + tmp.y * row * Zc;
			dev_buf[addr] = temp;
			
			
			if(threadIdx.x == thd && i == blk){
				printf("temp: %d, addr: %d\n", temp, addr);
			}
		}
	}
	__syncthreads();
}



__global__ void ldpc_decoder_gpu(int BG, char *dev_llr, char *dev_const_llr, char *dev_buf, int blk, int thd)
{
	int iteration = 5;
	int row = 46, col = 68;
	if(BG == 2){
		row = 42;
		col = 52;
	}
	//extern __shared__ sh_llr[];
	//for()
		
	for(int i = 0; i < iteration; i++){
		if(i == 0){
			cnp_1st(row, dev_llr, dev_buf, blk, thd);
		}else{
			cnp(row, dev_llr, dev_buf, blk, thd);
		}
		
		bnp(row, col, dev_llr, dev_const_llr, dev_buf, blk, thd);
	}
	
}


__global__ void pack_decoded_bit(char *dev, unsigned char *host)
{
	__shared__ unsigned char tmp[256];
	int tid = blockIdx.x*128 + threadIdx.x;
	int btid = threadIdx.x;
	tmp[btid] = 0;
	
	if(dev[tid] < 0){
		tmp[btid] = 1 << (7-(btid&7));
	}
	__syncthreads();
	
	if(threadIdx.x < 16){
		host[blockIdx.x*16+threadIdx.x] = 0;
		for(int i = 0; i < 8; i++){
			host[blockIdx.x*16+threadIdx.x] += tmp[threadIdx.x*8+i];
		}
	}
}


//void nrLDPC_decoder_LYC(int BG, int row, int col, int Zc, int block_length)
extern "C"
int32_t nrLDPC_decoder_LYC(t_nrLDPC_dec_params* p_decParams, int8_t* p_llr, int8_t* p_out, int block_length)
{
	// alloc mem
	unsigned char *decision = (unsigned char*)p_out;

    uint16_t Zc          = p_decParams->Z;
    uint8_t  BG         = p_decParams->BG;
    uint8_t  numMaxIter = p_decParams->numMaxIter;
    e_nrLDPC_outMode outMode = p_decParams->outMode;
	
	uint8_t row,col;
	if(BG == 1){
		row = 46;
		col = 68;
	}
	else{
		row = 42;
		col = 52;
	}
		// gpu mem size
	int memsize_llr = col * Zc * sizeof(char);
	int memsize_buf = col * Zc * row * sizeof(char);
	
	// gpu 
	char *dev_llr;
	char *dev_const_llr;
	char *dev_buf;
	unsigned char *dev_tmp;
	
	// gpu alloc
	int p;
	cudaCheck( hipMallocPitch((void**)&dev_llr, (size_t*)&p, memsize_llr, 1) );
	cudaCheck( hipMallocPitch((void**)&dev_const_llr, (size_t*)&p, memsize_llr, 1) );
	cudaCheck( hipMallocPitch((void**)&dev_buf, (size_t*)&p, memsize_buf, 1) );
	cudaCheck( hipMallocPitch((void**)&dev_tmp, (size_t*)&p, memsize_llr, 1) );
	
	// gpu memcpy
	cudaCheck( hipMemcpy((void*)dev_llr, p_llr, memsize_llr, hipMemcpyHostToDevice) );
	cudaCheck( hipMemcpy((void*)dev_const_llr, p_llr, memsize_llr, hipMemcpyHostToDevice) );
	//cudaCheck( hipMemcpy((void*)dev_const_llr, (const void*)p_llr, memsize_llr, hipMemcpyHostToDevice) );
	
	// gpu argument
	dim3 block(CW, 1, 1);
	dim3 thread(Zc, 1, 1);
	
	// debug info
	int blk, thd;
	fprintf(stderr, "block thread focus: \n");
	scanf("%d%d", &blk, &thd);
	
	ldpc_decoder_gpu<<<block, thread, memsize_llr>>>(BG, dev_llr, dev_const_llr, dev_buf, blk, thd);
	
	int pack = block_length/128;
	pack_decoded_bit<<<pack, 128>>>(dev_llr, dev_tmp);
	
	cudaCheck( hipMemcpy((void*)decision, (const void*)dev_tmp, block_length*sizeof(unsigned char), hipMemcpyDeviceToHost) );

	return MAX_ITERATION;
	
}
